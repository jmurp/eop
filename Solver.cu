#include "hip/hip_runtime.h"
#define PI 3.14159265358979323846
#include "Solver.h"

void Solver::calc_energy()
{
	reduce_energy<<<rblocks,nthreads,sizeof(double)*nthreads>>>(dNe, dRi, uK_n, vK_n, wK_n, bK_n, d_energy_reduce);
	cutilSafeCall( hipMemcpy(energy_reduce, d_energy_reduce, sizeof(double) * rblocks, hipMemcpyDeviceToHost) );
	energy_o = energy;
	energy = 0.0;
	for (int i = 0; i < rblocks; i++) {
		energy+= energy_reduce[i];
	}
	energy /= Ne;
	double sqrtenergy = sqrt(energy);
	cutilSafeCall( hipMemcpy(dsqrtenergy, &sqrtenergy, sizeof(double), hipMemcpyHostToDevice) );
};

void Solver::calc_residual()
{
	cutilSafeCall( hipMemcpy(uK_o, uR_o, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy(vK_o, vR_o, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy(wK_o, wR_o, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy(bK_o, bR_o, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyHostToDevice) );

	reduce_residual<<<rblocks,nthreads,sizeof(double)*nthreads>>>(dNe, dRi, uK_n, uK_o, vK_n, vK_o, wK_n, wK_o, bK_n, bK_o, d_residual_reduce);
	cutilSafeCall( hipMemcpy(residual_reduce, d_residual_reduce, sizeof(double) * rblocks, hipMemcpyDeviceToHost) );
	grad_residual = 0.0;
	for (int i = 0; i < rblocks; i++) {
		grad_residual+= residual_reduce[i];
	}
	grad_residual /= Ne*energy;
};

void Solver::optimize()
{
	init_IC<<<nblocks,nthreads>>>(dNe, ix, iy, iz, uK_n, vK_n, wK_n, bK_n);
	//calculates the energy of the solution (uK_n,vK_n,wK_n,bK_n) and normalize it
	energy = 0.0;
	calc_energy();
	normalize<<<nblocks,nthreads>>>(dNe, dsqrtenergy, uK_n, vK_n, wK_n, bK_n);
	//copy the normalized input back to uR_o as the first 'old' solution store
	cutilSafeCall( hipMemcpy(uR_o, uK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(vR_o, vK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(wR_o, wK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(bR_o, bK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );

	tolerance = 1.0e-6;
	grad_residual = tolerance + 1;
	energy_residual = tolerance + 1;
	int its = 0;

	clock_t start, end;
	double total;

	while (grad_residual > tolerance || energy_residual > tolerance) {

		//solve direct system, taking IC from uK_n and putting results (real space) to uK_n
		start = clock();
		direct_solve();
		end = clock();
		total = (double) (end - start) / CLOCKS_PER_SEC;
		printf("direct solve took %f seconds\n",total);

		refactor_b<<<nblocks,nthreads>>>(dNe, dRi, bK_n);

		//solve adjoint system, taking IC from uK_n and putting results (real space) to uK_n
		start = clock();
		adjoint_solve();
		end = clock();
		total = (double) (end - start) / CLOCKS_PER_SEC;
		printf("adjoint solve took %f seconds\n",total);

		//calulate new energy after storing old energy, normalize solution and calculate residual
		calc_energy();
		energy_residual = abs(energy - energy_o);
		normalize<<<nblocks,nthreads>>>(dNe, dsqrtenergy, uK_n, vK_n, wK_n, bK_n);
		calc_residual();

		//store the old solution
		cutilSafeCall( hipMemcpy(uR_o, uK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );
		cutilSafeCall( hipMemcpy(vR_o, vK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );
		cutilSafeCall( hipMemcpy(wR_o, wK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );
		cutilSafeCall( hipMemcpy(bR_o, bK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );

		std::cout << "\tSOLVER::OPTIMIZE::its = " << ++its << std::endl;
		std::cout << "\tSOVLER::OPTIMIZE::energy = " << energy << std::endl;
		std::cout << "\tSOLVER::OPTIMIZE::gradient residual = " << grad_residual << std::endl;
		std::cout << "\tSOLVER::OPTIMIZE::energy residual = " << energy_residual << std::endl;

		if (its > 10) break;

	}
};


int main() {

	int nblocks = 128;
	int nthreads = 512;
	int Nx = 64;
	int Ny = 128;
	int Nz = 32;
	double Lx = 2.0;
	double Ly = 4.0;
	double Lz = 2.0;
	double Re = 200.0;
	double Ri = 1.0;
	double Pr = 1.0;
	double T = 1.0;
	double dt = 0.01;

	Solver solver(nblocks,nthreads,Nx,Ny,Nz,Lx,Ly,Lz,Re,Ri,Pr,T,dt);

	solver.optimize();


};




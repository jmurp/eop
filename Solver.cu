#include "hip/hip_runtime.h"
#define PI 3.14159265358979323846
#include "Solver.h"

void Solver::calc_energy()
{
	reduce_energy<<<rblocks,nthreads,sizeof(double)*nthreads>>>(dNe, dRi, uK_n, vK_n, wK_n, bK_n, d_energy_reduce);
	cutilSafeCall( hipMemcpy(energy_reduce, d_energy_reduce, sizeof(double) * rblocks, hipMemcpyDeviceToHost) );
	energy_o = energy;
	energy = 0.0;
	for (int i = 0; i < rblocks; i++) energy+= energy_reduce[i];
	energy *= (0.5 / Ne);
	double sqrtenergy = sqrt(energy);
	cutilSafeCall( hipMemcpy(dsqrtenergy, &sqrtenergy, sizeof(double), hipMemcpyHostToDevice) );
};

void Solver::calc_residual()
{
	cutilSafeCall( hipMemcpy(uK_o, uR_o, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy(vK_o, vR_o, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy(wK_o, wR_o, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy(bK_o, bR_o, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyHostToDevice) );

	reduce_residual<<<rblocks,nthreads,sizeof(double)*nthreads>>>(dNe, dRi, uK_n, uK_o, vK_n, vK_o, wK_n, wK_o, bK_n, bK_o, d_residual_reduce);
	cutilSafeCall( hipMemcpy(residual_reduce, d_residual_reduce, sizeof(double) * rblocks, hipMemcpyDeviceToHost) );
	grad_residual = 0.0;
	for (int i = 0; i < rblocks; i++) grad_residual+= residual_reduce[i];
	reduce_residual_den<<<rblocks,nthreads,sizeof(double)*nthreads>>>(dNe, dRi, uK_n, vK_n, wK_n, bK_n, d_residual_reduce);
	cutilSafeCall( hipMemcpy(residual_reduce, d_residual_reduce, sizeof(double) * rblocks, hipMemcpyDeviceToHost) );
	double grad_residual_den = 0.0;
	for (int i = 0; i < rblocks; i++) grad_residual_den+= residual_reduce[i];

	grad_residual /= Ne*energy;//old -> lead to decreasing gradient residual
	//grad_residual /= grad_residual_den;//new -> gradient residual does not change

	//std::cout << "Ne*energy = " << Ne*energy << std::endl;
	//std::cout << "grad_residual_den = " << grad_residual_den << std::endl;
};

void Solver::optimize()
{
	log_begin_optimize();
	init_data_files();

	init_IC<<<nblocks,nthreads>>>(dNe, ix, iy, iz, uK_n, vK_n, wK_n, bK_n);
	//calculates the energy of the solution (uK_n,vK_n,wK_n,bK_n) and normalize it
	energy = 0.0;
	calc_energy();
	normalize<<<nblocks,nthreads>>>(dNe, dsqrtenergy, uK_n, vK_n, wK_n, bK_n);
	//copy the normalized input back to uR_o as the first 'old' solution store
	cutilSafeCall( hipMemcpy(uR_o, uK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(vR_o, vK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(wR_o, wK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(bR_o, bK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );

	tolerance = 1.0e-6;
	grad_residual = tolerance + 1;
	energy_residual = tolerance + 1;
	int its = 0;

	clock_t optimize_start, optimize_end;
	clock_t solver_start, solver_end;
	double direct_time = 0.0;
	double adjoint_time = 0.0; 
	double optimize_time = 0.0;

	optimize_start = clock();
	while (grad_residual > tolerance) {

		//solve direct system, taking IC from uK_n and putting results (real space) to uK_n
		solver_start = clock();
		direct_solve();
		solver_end = clock();
		direct_time += (double) (solver_end - solver_start) / CLOCKS_PER_SEC;

		//store gain, kinetic, potential, and other
		write_to_data_files();

		refactor_b<<<nblocks,nthreads>>>(dNe, dRi, bK_n);

		//solve adjoint system, taking IC from uK_n and putting results (real space) to uK_n
		solver_start = clock();
		adjoint_solve();
		solver_end = clock();
		adjoint_time += (double) (solver_end - solver_start) / CLOCKS_PER_SEC;

		//calulate new energy after storing old energy, normalize solution and calculate residual
		calc_energy();
		energy_residual = abs(energy - energy_o) / energy;
		normalize<<<nblocks,nthreads>>>(dNe, dsqrtenergy, uK_n, vK_n, wK_n, bK_n);
		calc_residual();

		//store the old solution
		cutilSafeCall( hipMemcpy(uR_o, uK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );
		cutilSafeCall( hipMemcpy(vR_o, vK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );
		cutilSafeCall( hipMemcpy(wR_o, wK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );
		cutilSafeCall( hipMemcpy(bR_o, bK_n, sizeof(hipfftDoubleComplex) * Ne, hipMemcpyDeviceToHost) );

		Solver::log("\nSOLVER::OPTIMIZE::its = " + std::to_string((++its)));
		Solver::log("SOLVER::OPTIMIZE::energy = " + std::to_string(energy));
		Solver::log("SOLVER::OPTIMIZE::gradient residual = " + std::to_string(grad_residual));
		Solver::log("SOLVER::OPTIMIZE::energy residual = " + std::to_string(energy_residual) + "\n");

	}
	optimize_end = clock();
	optimize_time = (double) (optimize_end - optimize_start) / CLOCKS_PER_SEC;

	Solver::log("\nSOLVER::OPTIMIZE::DONE::its = " + std::to_string(its));
	Solver::log("SOLVER::OPTIMIZE::DONE::energy = " + std::to_string(energy));
	Solver::log("SOLVER::OPTIMIZE::DONE::gradient_residual = " + std::to_string(grad_residual));
	Solver::log("SOLVER::OPTIMIZE::DONE::energy residual = " + std::to_string(energy_residual));
	Solver::log("\n\tSOLVER::average time for direct_solve() = " + std::to_string(( (double) direct_time / its )));
	Solver::log("\tSOLVER::average time for adjoint_solve() = " + std::to_string(( (double) adjoint_time / its )));
	Solver::log("\tSOLVER::optimize time elapsed = " + std::to_string(optimize_time));

	close_data_files();

};


int main() {

	double Ly_arr[3] = {2.0, 4.0, 6.0};

	int nblocks = 128;
	int nthreads = 512;
	int Nx = 64;
	int Ny = 128;
	int Nz = 32;
	double Lx = 2.0;
	double Ly = 4.0;
	double Lz = 2.0;
	double Re = 100.0;
	double Ri = 1.0;
	double Pr = 1.0;
	double T = 1.0;
	double dt = 0.01;

	Solver solver(nblocks,nthreads,Nx,Ny,Nz,Lx,Ly_arr[0],Lz,Re,Ri,Pr,T,dt);

	for (int i = 0; i < 3; i++) {

		solver.optimize();

		if (i != 2) solver.reset(nblocks,nthreads,Nx,Ny,Nz,Lx,Ly_arr[i+1],Lz,Re,Ri,Pr,T,dt);
	}

};



